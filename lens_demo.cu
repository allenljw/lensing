#include "hip/hip_runtime.h"
/* 
   159735 Parallel Programming

   Startup program for sequential implementation of simulation by ray
   tracing of gravitational lensing.
 */
#include <ctime>

#include <iostream>
#include <string>
#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>

#include "lenses.h"
#include "arrayff.hxx"


// Global variables! Not nice style, but we'll get away with it here.
#define BLOCKSIZE_x 32
#define BLOCKSIZE_y 32

// Boundaries in physical units on the lens plane
const float WL  = 2.0;
const float XL1 = -WL;
const float XL2 =  WL;
const float YL1 = -WL;
const float YL2 =  WL;


// Implement lens equation, given the lens position (xl, yl) and the
// lens system configuration, shoot a ray back to the source position
// (xs, ys)
__device__ void shoot(float& xs, float& ys, float xl, float yl, 
  float* xlens, float* ylens, float* eps, int nlenses)
{
  float dx, dy, dr;
  xs = xl;
  ys = yl;
  for (int p = 0; p < nlenses; ++p) {
    dx = xl - xlens[p];
    dy = yl - ylens[p];
    dr = dx * dx + dy * dy;
    xs -= eps[p] * dx / dr;
    ys -= eps[p] * dy / dr;
  }

}


// Used to time code. OK for single threaded programs but not for
// multithreaded programs. See other demos for hints at timing CUDA
// code.
double diffclock(clock_t clock1,clock_t clock2)
{
  double diffticks = clock1 - clock2;
  double diffms = (diffticks * 1000) / CLOCKS_PER_SEC;
  return diffms; // Time difference in milliseconds
}

__global__ void mx_shoot(float* xlens, float* ylens, float* eps, float* d_lensim, float XL1, float YL1, int nlenses, float lens_scale) 
{
    // Source star parameters. You can adjust these if you like - it is
    // interesting to look at the different lens images that result
    const float rsrc = 0.1;      // radius
    const float ldc = 0.5;      // limb darkening coefficient
    const float xsrc = 0.0;      // x and y centre on the map
    const float ysrc = 0.0;
    const float rsrc2 = rsrc * rsrc;

    float xl, yl, xs, ys, sep2, mu;
    float xd, yd;

    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;


    yl = YL1 + row * lens_scale;
    xl = XL1 + col * lens_scale;

    shoot(xs, ys, xl, yl, xlens, ylens, eps, nlenses);

    std::cout << "device col:" << col << ", row:" << row << std::endl;
    // xd = xs - xsrc;
    // yd = ys - ysrc;
    // sep2 = xd * xd + yd * yd;
    // if (sep2 < rsrc2) {
    //     mu = sqrt(1 - sep2 / rsrc2);
    //     lensim(row, col) = 1.0 - ldc * (1 - mu);
    // }

}

int iDivUp(int hostPtr, int b){ return ((hostPtr % b) != 0) ? (hostPtr / b + 1) : (hostPtr / b); }

int main(int argc, char* argv[]) 
{
  // Set up lensing system configuration - call example_1, _2, _3 or
  // _n as you wish. The positions and mass fractions of the lenses
  // are stored in these arrays
  float* xlens;
  float* ylens;
  float* eps;
  const int nlenses = set_example_3(&xlens, &ylens, &eps);
  std::cout << "# Simulating " << nlenses << " lens system" << std::endl;



  // Pixel size in physical units of the lens image. You can try finer
  // lens scale which will result in larger images (and take more
  // time).
  const float lens_scale = 0.005;

  // Size of the lens image
  const int npixx = static_cast<int>(floor((XL2 - XL1) / lens_scale)) + 1;
  const int npixy = static_cast<int>(floor((YL2 - YL1) / lens_scale)) + 1;
  std::cout << "# Building " << npixx << "X" << npixy << " lens image" << std::endl;

  // Put the lens image in this array
  Array<float, 2> lensim(npixy, npixx);

  clock_t tstart = clock();

  //declare the variables for device function here
  // copy the host variables to device variables
  //float* d_xlens, d_ylens, d_eps, d_lensim;
  size_t size = nlenses * sizeof(float);
  size_t pitch;

  // hipMalloc(&d_xlens, size);
  // hipMalloc(&d_ylens, size);
  // hipMalloc(&d_eps, size);

  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);


  // hipMemcpy(d_xlens, xlens, size, hipMemcpyHostToDevice);
  // hipMemcpy(d_ylens, ylens, size, hipMemcpyHostToDevice);
  // hipMemcpy(d_eps, eps, size, hipMemcpyHostToDevice);

  // hipMallocPitch(&d_lensim, &pitch, npixx * sizeof(float), npixy);
  // hipMemcpy2D(d_lensim, pitch, lensim, npixx*sizeof(float), npixx*sizeof(float), npixy, hipMemcpyHostToDevice);

  // //use the device function here
  // dim3 gridSize(iDivUp(npixx, BLOCKSIZE_x), iDivUp(npixy, BLOCKSIZE_y));
  // dim3 blockSize(BLOCKSIZE_y, BLOCKSIZE_x);

  // mx_shoot<<<gridSize, blockSize>>>(d_xlens, d_ylens, d_eps, d_lensim, XL1, YL1, nlenses, lens_scale);


  // Draw the lensing image map here. For each pixel, shoot a ray back
  // to the source plane, then test whether or or not it hits the
  // source star
  // replace with device function
  /*const float rsrc2 = rsrc * rsrc;
  float xl, yl, xs, ys, sep2, mu;
  float xd, yd;
  int numuse = 0;
  for (int iy = 0; iy < npixy; ++iy) 
  for (int ix = 0; ix < npixx; ++ix) { 

    yl = YL1 + iy * lens_scale;
    xl = XL1 + ix * lens_scale;

    shoot(xs, ys, xl, yl, xlens, ylens, eps, nlenses);

    xd = xs - xsrc;
    yd = ys - ysrc;
    sep2 = xd * xd + yd * yd;
    if (sep2 < rsrc2) {
      mu = sqrt(1 - sep2 / rsrc2);
      lensim(iy, ix) = 1.0 - ldc * (1 - mu);
    }
  }*/

  clock_t tend = clock();
  double tms = diffclock(tend, tstart);
  std::cout << "# Time elapsed: " << tms << " ms " << std::endl;

  // Write the lens image to a FITS formatted file. You can view this
  // image file using ds9
  dump_array<float, 2>(lensim, "lens.fit");

  delete[] xlens;
  delete[] ylens;
  delete[] eps;

  // hipFree( d_xlens );
  // hipFree( d_ylens );
  // hipFree( d_eps );
}

